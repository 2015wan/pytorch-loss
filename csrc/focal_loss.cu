#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

#include <torch/extension.h>



template<typename scalar_t>
__global__ void FocalLossForward(const int nthreads,
                                 const scalar_t *logits,
                                 const int64_t *labels,
                                 scalar_t *loss,
                                 const scalar_t gamma, const scalar_t alpha) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    const scalar_t one(1.);
    for (int i{tid}; i < nthreads; i+=stride) {
        scalar_t lgt = logits[i];
        scalar_t prob = one / (one + expf(-lgt));
        scalar_t log_p, log_1_p;
        if (lgt >= 0) {
            log_p = -logf(one + expf(-lgt));
            log_1_p = -lgt + log_p;
        } else {
            log_1_p = -logf(one + expf(lgt));
            log_p = lgt + log_1_p;
        }
        scalar_t term1 = powf(one - prob, gamma) * log_p;
        scalar_t term2 = powf(prob, gamma) * log_1_p;
        loss[i] = -alpha * term1 * labels[i] - (one - alpha) * term2 * (one - labels[i]);
    }
}

template<typename scalar_t>
__global__ void FocalLossBackward(const int nthreads,
                                  const scalar_t *logits,
                                  const int64_t *labels,
                                  const scalar_t *grad_loss,
                                  scalar_t *grad_logits,
                                  const scalar_t gamma, const scalar_t alpha) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    const scalar_t one(1.);
    for (int i{tid}; i < nthreads; i+=stride) {
        scalar_t lgt = logits[i];
        scalar_t prob = one / (one + expf(-lgt));
        scalar_t log_p, log_1_p;
        if (lgt >=0) {
            log_p = -logf(one + expf(-lgt));
            log_1_p = -lgt + log_p;
        } else {
            log_1_p = -logf(one + expf(lgt));
            log_p = lgt + log_1_p;
        }
        scalar_t term1 = powf(one - prob, gamma) * (one - prob - gamma * prob * log_p);
        scalar_t term2 = powf(prob, gamma) * (gamma * (one - prob) * log_1_p - prob);
        grad_logits[i] = -alpha * term1 * labels[i] - (one - alpha) * term2 * (one - labels[i]);
        grad_logits[i] = grad_logits[i] * grad_loss[i];
    }
}


at::Tensor FocalLoss_forward_cuda(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float gamma,
                                  const float alpha) {
    // CHECK type and shape
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");

    // allocate memory and cuda grid/block
    auto losses = at::empty_like(logits);

    const int num_samples = logits.numel();
    dim3 grid(std::min(
        THCCeilDiv((int64_t)num_samples, (int64_t)512), (int64_t)4096
    ));
    dim3 block(512);
    if (losses.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return losses;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(logits.scalar_type(), "focal forward", [&] {
        FocalLossForward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(),
            losses.contiguous().data<scalar_t>(),
            scalar_t(gamma), scalar_t(alpha)
        );
    });
    THCudaCheck(hipGetLastError());
    return losses;
}


at::Tensor FocalLoss_backward_cuda(const at::Tensor &grad,
                                  const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float gamma,
                                  const float alpha) {
    // CHECK type and shape
    AT_ASSERTM(grad.type().is_cuda(), "grad should be cuda");
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");
    // AT_ASSERTM(logits.size() == labels.size(), "should have same shape");

    /* allocate memory and cuda grid/block */
    auto grad_logits = at::empty_like(logits);
    const int num_samples = logits.numel();
    dim3 grid(std::min(
        THCCeilDiv((int64_t)num_samples, (int64_t)512), (int64_t)4096
    ));
    dim3 block(512);
    if (grad_logits.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return grad_logits;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(logits.scalar_type(), "focal backwrd", [&] {
        FocalLossBackward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(),
            grad.contiguous().data<scalar_t>(),
            grad_logits.contiguous().data<scalar_t>(),
            scalar_t(gamma), scalar_t(alpha)
        );
    });
    THCudaCheck(hipGetLastError());
    return grad_logits;
}

// python inferface
at::Tensor FocalLoss_forward(const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float gamma,
                             const float alpha) {
    if (!logits.type().is_cuda() || !labels.type().is_cuda()) {
        AT_ERROR("this focal loss only support gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return FocalLoss_forward_cuda(logits, labels, gamma, alpha);
}

at::Tensor FocalLoss_backward(const at::Tensor &grad,
                             const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float gamma,
                             const float alpha) {
    // TODO: try AT_ASSERTM
    if (!logits.type().is_cuda()) {
        AT_ERROR("this focal loss only support gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return FocalLoss_backward_cuda(grad, logits, labels, gamma, alpha);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("focalloss_forward", &FocalLoss_forward, "focal loss forward");
    m.def("focalloss_backward", &FocalLoss_backward, "focal loss backward");
}
