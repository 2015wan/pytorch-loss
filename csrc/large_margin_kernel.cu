#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>

#include <iostream>

using std::cout;
using std::endl;

#define BLOCKSIZE 512

// TODO: 
// 1. gridDim consider bother shm and sample numbers
// 2. all loss change into compute gradient product outsize of kernel, do it in python side

// kernel function for forward and backward
template<typename scalar_t>
__global__ void LMarginLossForward(const int n_size,
                            const int dimsize, const int m_size,
                            const scalar_t *logits,
                            const int64_t *labels,
                            scalar_t *losses,
                            const int64_t ignore_index, const float lam) {
    // shared memory
    // b is max logits without target 
    // b+1 is max logits with target 
    // b+2 is sum of exp without target 
    // b+3 is sum of exp with target 
    extern __shared__ __align__(sizeof(scalar_t)) unsigned char sdata_raw[];
    scalar_t *sdata = reinterpret_cast<scalar_t*>(sdata_raw);

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    scalar_t coeff = 1. / (dimsize - 1);


    // int tar = 10 * m_size + 1 * 16 + 12;
    // if (bid == 0 && tid == 0) {
    //     printf("%d, \n", tar);
    // }

    int samplesize = n_size * m_size;
    for (int i{bid}; i < samplesize; i+=gridDim.x) {
        int64_t lb = labels[i];
        if (lb == ignore_index) {
            if (tid == 0) losses[i] = 0;
            continue;
        } 
        int n_idx = i / m_size;
        int m_idx = i % m_size;

        // compute max value for each vector for softmax
        sdata[tid] = -1000;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            if (j == lb) continue;
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            if (dval > sdata[tid]) sdata[tid] = dval;
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                if (sdata[idx] < sdata[idx + s]) sdata[idx] = sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x] = sdata[0]; // max logits without label
            sdata[blockDim.x + 1] = sdata[0]; // max logits with label
            int idx = n_idx * dimsize * m_size + lb * m_size + m_idx; 
            scalar_t dval = logits[idx];
            if (dval > sdata[0]) sdata[blockDim.x + 1] = dval;
        }

        // compute exp sum for softmax
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            if (j == lb) continue;
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            sdata[tid] += expf(dval - sdata[blockDim.x]);
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x + 2] = sdata[0]; // exp sum without label
        }
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            sdata[tid] += expf(dval - sdata[blockDim.x + 1]);
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x + 3] = sdata[0]; // exp sum with label
        }

        // if (i == tar && tid == 0) {
        //     printf("sdata: ");
        //     for (int ii{0}; ii < 4; ++ii) {
        //         printf("%lf, ", sdata[blockDim.x + ii]);
        //     }
        //     printf("\n");
        //     printf("logits:");
        //     for (int ii{0}; ii < dimsize; ++ii) {
        //         int idx = n_idx * dimsize * m_size + ii * m_size + m_idx;
        //         scalar_t dval = logits[idx];
        //         printf("%lf, ", dval);
        //     }
        //     printf("\n");
        // }

        // compute extra term
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            scalar_t term{0};
            if (j == lb) {
                term = -(dval - sdata[blockDim.x + 1]);
                term += logf(sdata[blockDim.x + 3]);
            } else {
                dval -= sdata[blockDim.x];
                term = expf(dval) / sdata[blockDim.x + 2] - coeff;
                term *= (dval - logf(sdata[blockDim.x + 2]));
                term *= lam / 2.;
            }
            sdata[tid] += term;
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) losses[i] = sdata[0];
    }
}


template<typename scalar_t>
__global__ void LMarginLossBackward(const int n_size,
                            const int dimsize, const int m_size,
                            scalar_t *grad_logits,
                            const scalar_t *logits,
                            const int64_t *labels,
                            const int64_t ignore_index,
                            const float lam) {
    extern __shared__ __align__(sizeof(scalar_t)) unsigned char sdata_raw[];
    scalar_t *sdata = reinterpret_cast<scalar_t*>(sdata_raw);
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    scalar_t coeff = 1. / (dimsize - 1);

    // int tar = 3 * m_size + 1 * 16 + 2;
    // if (bid == 0 && tid == 0) {
    //     printf("%d, \n", tar);
    // }

    int samplesize = n_size * m_size;
    for (int i{bid}; i < samplesize; i+=gridDim.x) {
        int64_t lb = labels[i];
        int n_idx = i / m_size;
        int m_idx = i % m_size;
        if (lb == ignore_index) {
            for (int j{tid}; j < dimsize; j+=blockDim.x) {
                int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
                grad_logits[idx] = 0;
            }
            continue;
        } 

        // compute max value for each vector for softmax
        sdata[tid] = -1000;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            if (j == lb) continue;
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            if (dval > sdata[tid]) sdata[tid] = dval;
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                if (sdata[idx] < sdata[idx + s]) sdata[idx] = sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x] = sdata[0]; // max logits without label
            sdata[blockDim.x + 1] = sdata[0]; // max logits with label
            int idx = n_idx * dimsize * m_size + lb * m_size + m_idx; 
            scalar_t dval = logits[idx];
            if (dval > sdata[0]) sdata[blockDim.x + 1] = dval;
        }

        // compute exp sum for softmax
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            if (j == lb) continue;
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            sdata[tid] += expf(dval - sdata[blockDim.x]);
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x + 2] = sdata[0]; // exp sum without label
        }
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            sdata[tid] += expf(dval - sdata[blockDim.x + 1]);
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x + 3] = sdata[0]; // exp sum with label
        }

        // if (i == tar && tid == 0) {
        //     printf("sdata: ");
        //     for (int ii{0}; ii < 4; ++ii) {
        //         printf("%lf, ", sdata[blockDim.x + ii]);
        //     }
        //     printf("\n");
        //     printf("logits:");
        //     for (int ii{0}; ii < dimsize; ++ii) {
        //         int idx = n_idx * dimsize * m_size + ii * m_size + m_idx;
        //         scalar_t dval = logits[idx];
        //         printf("%lf, ", dval);
        //     }
        //     printf("\n");
        // }

        // compute sum of q * x
        sdata[tid] = 0.;
        __syncthreads();
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            if (j == lb) continue;
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            scalar_t tmp = dval * expf(dval - sdata[blockDim.x]);
            sdata[tid] += tmp;
            // if (i == tar) {
            //     if (tid == 0) printf("qx: ");
            //     printf("%f, ", tmp / sdata[blockDim.x + 2]);
            //     if (tid == 0) printf("\n ");
            // }
        }
        __syncthreads();
        for (int s=1; s < blockDim.x; s*=2) {
            int idx = 2 * s * threadIdx.x;
            if (idx < blockDim.x && idx + s < blockDim.x) {
                sdata[idx] += sdata[idx + s];
            }
            __syncthreads();
        }
        if (tid == 0) {
            sdata[blockDim.x + 4] = sdata[0] / sdata[blockDim.x + 2]; 
            // if (i == tar && tid == 0)
            // printf("\nsum of qx: %f\n", sdata[blockDim.x + 4]);
        }
        for (int j{tid}; j < dimsize; j+=blockDim.x) {
            int idx = n_idx * dimsize * m_size + j * m_size + m_idx; 
            scalar_t dval = logits[idx];
            scalar_t pc = expf(dval - sdata[blockDim.x + 1]) / sdata[blockDim.x + 3];
            // if (i == tar) {
            //     if (tid == 0) printf("pc, and dval: ");
            //     printf("%f, ", pc);
            //     printf("%f, ", dval);
            //     if (tid == 0) printf("\n ");
            // }
            scalar_t gval;
            if (j == lb) {
                gval = pc - 1.;
            } else {
                gval = dval - sdata[blockDim.x + 4] + 1.;
                // if (i == tar) {
                //     if (tid == 2) printf("gval: ");
                //     printf("%f, ", gval);
                //     if (tid == 2) printf("\n ");
                // }
                gval *= expf(dval - sdata[blockDim.x]) / sdata[blockDim.x + 2]; 
                // if (i == tar) {
                //     if (tid == 2) printf("gval: ");
                //     printf("%f, ", gval);
                //     if (tid == 2) printf("\n ");
                // }
                gval = pc + (gval - coeff) * lam / 2.;
                // if (i == tar) {
                //     if (tid == 2) printf("gval: ");
                //     printf("%f, ", gval);
                //     if (tid == 2) printf("\n ");
                // }
            }
            // if (i == tar) {
            //     if (tid == 2) printf("idx: ");
            //     printf("%d, ", idx);
            //     if (tid == 2) printf("\n ");
            // }
            // if (i == tar) {
            //     if (tid == 2) printf("grad[idx]: ");
            //     printf("%f, ", grad[idx]);
            //     if (tid == 2) printf("\n ");
            // }
            // if (i == tar) {
            //     if (tid == 2) printf("grad_logits[i]: ");
            //     printf("%f, ", grad_logits[i]);
            //     if (tid == 2) printf("\n ");
            // }
            grad_logits[idx] = gval;
            // sdata[tid] += dval * expf(dval - sdata[blockDim.x]);
            // if (i == 0 && tid == 0) printf("\n gval: ");
            // if (i == 0) {
            //     printf("%f, ", gval);
            // }
            // if (i == 0 && tid == 0) printf("\n grad_output: " );
            // if (i == 0 && tid == 0) {
            //     printf("%f, ", grad[idx]);
            // }
            // if (i == 0 && tid == 0) printf("\n");
        }

    }
}


// cuda forward and backward
at::Tensor large_margin_forward_cuda(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const int64_t ignore_index,
                                  const float lam) {
    // CHECK type and shape
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");

    const int n_size = logits.size(0);
    const int dimsize = logits.size(1);
    const int m_size = logits.numel() / (n_size * dimsize);
    const int samplesize = labels.numel();

    // allocate memory and cuda grid/block
    auto losses = torch::empty_like(labels, logits.options());
    if (losses.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return losses;
    }


    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(losses.scalar_type(), "large margin forward", [&] {
        int blockdim = 32;
        if (dimsize > 32) blockdim = 64;
        dim3 block(blockdim);
        int griddim = 48 * 1024 / sizeof(scalar_t) / blockdim;
        dim3 grid(std::min(griddim, (int)samplesize));
        int shm_size = (blockdim + 6) * sizeof(scalar_t); 
        LMarginLossForward<scalar_t><<<grid, block, shm_size, at::cuda::getCurrentCUDAStream()>>>(
            n_size, dimsize, m_size, 
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(), 
            losses.contiguous().data<scalar_t>(),
            ignore_index, lam 
        );
    });
    THCudaCheck(hipGetLastError());
    return losses;
}


at::Tensor large_margin_backward_cuda(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const int64_t ignore_index,
                                  const float lam) {
    // CHECK type and shape
    AT_ASSERTM(logits.type().is_cuda(), "logits should be cuda");
    AT_ASSERTM(labels.type().is_cuda(), "labels should be cuda");

    const int n_size = logits.size(0);
    const int dimsize = logits.size(1);
    const int m_size = logits.numel() / (n_size * dimsize);
    const int samplesize = labels.numel();

    // allocate memory and cuda grid/block
    auto grad_logits = torch::empty_like(logits);
    if (grad_logits.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return grad_logits;
    }


    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_logits.scalar_type(), "large margin backwrd", [&] {
        int blockdim = 32;
        if (dimsize > 32) blockdim = 64;
        dim3 block(blockdim);
        int griddim = 48 * 1024 / sizeof(scalar_t) / blockdim;
        dim3 grid(std::min(griddim, (int)samplesize));
        int shm_size = (blockdim + 6) * sizeof(scalar_t); 
        LMarginLossBackward<scalar_t><<<grid, block, shm_size, at::cuda::getCurrentCUDAStream()>>>(
            n_size, dimsize, m_size, 
            grad_logits.contiguous().data<scalar_t>(),
            logits.contiguous().data<scalar_t>(), 
            labels.contiguous().data<int64_t>(), 
            ignore_index,lam 
        );
    });
    THCudaCheck(hipGetLastError());
    return grad_logits;
}

// python inferface
at::Tensor large_margin_forward(const at::Tensor &logits,
                             const at::Tensor &labels,
                             const float lam,
                             const int64_t ignore_index) {
    if (!(logits.type().is_cuda() && labels.type().is_cuda())) {
        AT_ERROR("this large margin loss only supports gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return large_margin_forward_cuda(logits, labels, ignore_index, lam);
}


at::Tensor large_margin_backward(const at::Tensor &logits,
                                  const at::Tensor &labels,
                                  const float lam,
                                  const int64_t ignore_index) {
    // TODO: try AT_ASSERTM
    if (!(logits.type().is_cuda() && labels.type().is_cuda())) {
        AT_ERROR("this large margin loss only supports gpu mode\n");
    } 
    at::DeviceGuard guard(logits.device());
    return large_margin_backward_cuda(logits, labels, ignore_index, lam);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("l_margin_forward", &large_margin_forward, "large margin forward");
    m.def("l_margin_backward", &large_margin_backward, "large margin backward");
}
