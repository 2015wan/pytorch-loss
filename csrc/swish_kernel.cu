#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cfloat>


#define BLOCKSIZE 1024


// NOTE: If use constant number such as 1. or 2., must use scalar_t(1.) or scalar_t(2.), or the values will be casted into double type.

// kernel function for forward and backward
template<typename scalar_t>
__global__ void SwishForward(const int nthreads,
                            const scalar_t *feat,
                            scalar_t *activations) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i{tid}; i < nthreads; i+=stride) {
        const scalar_t one(1.);
        scalar_t val = feat[i];
        activations[i] = val / (one + expf(-val));
    }
}

template<typename scalar_t>
__global__ void SwishBackward(const int nthreads,
                             const scalar_t *feat,
                             const scalar_t *grad,
                             scalar_t *grad_feat) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i{tid}; i < nthreads; i+=stride) {
        const scalar_t one(1.);
        scalar_t val = feat[i];

        grad_feat[i] = (one + val / (one + expf(val))) / (one + expf(-val));
        grad_feat[i] *= grad[i];

    }
}


// cuda forward and backward
at::Tensor Swish_forward_cuda(const at::Tensor &feat) {
    // CHECK type and shape
    AT_ASSERTM(feat.type().is_cuda(), "feat should be cuda");

    // allocate memory and cuda grid/block
    auto activations = at::empty_like(feat);

    const int num_samples = feat.numel();
    dim3 grid(std::min(
        THCCeilDiv(num_samples, 2 * BLOCKSIZE), 4096
    ));
    dim3 block(BLOCKSIZE);
    if (activations.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return activations;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(activations.scalar_type(), "swish forward", [&] {
        SwishForward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            feat.contiguous().data<scalar_t>(), 
            activations.contiguous().data<scalar_t>()
        );
    });
    THCudaCheck(hipGetLastError());
    return activations;
}


at::Tensor Swish_backward_cuda(const at::Tensor &grad, const at::Tensor &feat) {
    // CHECK type and shape
    AT_ASSERTM(grad.type().is_cuda(), "grad should be cuda");
    AT_ASSERTM(feat.type().is_cuda(), "feat should be cuda");

    // allocate memory and cuda grid/block
    auto grad_feat = at::empty_like(feat);

    const int num_samples = feat.numel();
    dim3 grid(std::min(
        // THCCeilDiv(num_samples, BLOCKSIZE), 4096
        THCCeilDiv(num_samples, 2 * BLOCKSIZE), 4096
    ));
    dim3 block(BLOCKSIZE);
    if (grad_feat.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return grad_feat;
    }

    // call kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_feat.scalar_type(), "swish backwrd", [&] {
        SwishBackward<scalar_t><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_samples, 
            feat.contiguous().data<scalar_t>(), 
            grad.contiguous().data<scalar_t>(),
            grad_feat.contiguous().data<scalar_t>()
        );
    });
    THCudaCheck(hipGetLastError());
    return grad_feat;
}

// python inferface
at::Tensor Swish_forward(const at::Tensor &feat) {
    if (!feat.type().is_cuda()) {
        AT_ERROR("this swish function only supports gpu mode\n");
    } 
    at::DeviceGuard guard(feat.device());
    return Swish_forward_cuda(feat);
}

at::Tensor Swish_backward(const at::Tensor &grad, const at::Tensor &feat) {
    // TODO: try AT_ASSERTM
    if (!feat.type().is_cuda()) {
        AT_ERROR("this swish function only supports gpu mode\n");
    } 
    at::DeviceGuard guard(feat.device());
    return Swish_backward_cuda(grad, feat);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("swish_forward", &Swish_forward, "swish forward");
    m.def("swish_backward", &Swish_backward, "swish backward");
}
